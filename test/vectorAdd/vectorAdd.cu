#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

 #include <stdio.h>

 // For the CUDA runtime routines (prefixed with "cuda_")
 #include <hip/hip_runtime.h>
 #include "allocator.h"
 /**
  * CUDA Kernel Device code
  *
  * Computes the vector addition of A and B into C. The 3 vectors have the same
  * number of elements numElements.
  */
 __global__ void
 vectorAdd(const float *A, const float *B, float *C, int numElements)
 {
     int i = blockDim.x * blockIdx.x + threadIdx.x;
 
     if (i < numElements)
     {
         C[i] = A[i] + B[i];
     }
 }
 
 /**
  * Host main routine
  */
 int
 main(void)
 {
     // Print the vector length to be used, and compute its size
     int numElements = 50000;
     size_t size = numElements * sizeof(float);
     printf("[Vector addition of %d elements]\n", numElements);
 
     // Allocate the host input vector A
    //  float *h_A = (float *)malloc(size);
    auto h_A = MemAlloc(size, CPUNOPINNOMAP);

 
     // Allocate the host input vector B
    //  float *h_B = (float *)malloc(size);
     auto h_B = MemAlloc(size, CPUNOPINNOMAP);
 
     // Allocate the host output vector C
    //  float *h_C = (float *)malloc(size);
     auto h_C = MemAlloc(size, CPUNOPINNOMAP);
 
     // Verify that allocations succeeded
     if(h_A.err != UM_SUCCESS) {
         fprintf(stderr, "fail to allocate memory");
         exit(EXIT_FAILURE);
     }
 
     // Initialize the host input vectors
     for (int i = 0; i < numElements; ++i)
     {
         h_A.uptr.cpu_address[i] = rand()/(float)RAND_MAX;
         h_B.uptr.cpu_address[i] = rand()/(float)RAND_MAX;
     }
 
     // Allocate the device input vector A
     float *d_A = NULL;
    auto  err = hipMalloc((void **)&d_A, size);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     // Allocate the device input vector B
     float *d_B = NULL;
     err = hipMalloc((void **)&d_B, size);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     // Allocate the device output vector C
     float *d_C = NULL;
     err = hipMalloc((void **)&d_C, size);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     // Copy the host input vectors A and B in host memory to the device input vectors in
     // device memory
     printf("Copy input data from the host memory to the CUDA device\n");
     err = hipMemcpy(d_A, h_A.uptr.cpu_address, size, hipMemcpyHostToDevice);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     err = hipMemcpy(d_B, h_B.uptr.cpu_address, size, hipMemcpyHostToDevice);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     // Launch the Vector Add CUDA Kernel
     int threadsPerBlock = 256;
     int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
     printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
     vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
     err = hipGetLastError();
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     // Copy the device result vector in device memory to the host result vector
     // in host memory.
     printf("Copy output data from the CUDA device to the host memory\n");
     err = hipMemcpy(h_C.uptr.cpu_address, d_C, size, hipMemcpyDeviceToHost);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     // Verify that the result vector is correct
     for (int i = 0; i < numElements; ++i)
     {
         if (fabs(h_A.uptr.cpu_address[i] + h_B.uptr.cpu_address[i] - h_C.uptr.cpu_address[i]) > 1e-5)
         {
             fprintf(stderr, "Result verification failed at element %d!\n", i);
             exit(EXIT_FAILURE);
         }
     }
 
     printf("Test PASSED\n");
 
     // Free device global memory
     err = hipFree(d_A);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     err = hipFree(d_B);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     err = hipFree(d_C);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     // Free host memory
     h_A.uptr.free();
     h_B.uptr.free();
     h_C.uptr.free();
 
     printf("Done\n");
     return 0;
 }
 