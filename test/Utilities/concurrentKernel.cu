//
// Created by mark on 2021/3/2.
//

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int main() {
    int nKernels = 8;
    int nStreams = nKernels + 1;
    int nbytes = nKernels * sizeof(clock_t);
    float kernel_time = 10;
    float elapsed_time;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, 0);
    if(deviceProp.concurrentKernels != 0) {
        cout << "GPU does not support concurrent kernels execution" << endl;
        cout << "CUDA kernel runs will be serialized\n" << endl;
    }
    cout << deviceProp.asyncEngineCount << endl;
    clock_t *a = nullptr;
    hipHostMalloc((void **)&a, nbytes, hipHostMallocDefault);
    clock_t *d_a = nullptr;
    hipMalloc((void **)&d_a, nbytes);

    auto streams = (hipStream_t *)malloc(nStreams * sizeof(hipStream_t));

    for(int i = 0; i < nStreams; i++) {
        hipStreamCreate(&streams[i]);
    }
    hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, streams[1]);
}