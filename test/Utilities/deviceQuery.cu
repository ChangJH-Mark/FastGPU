//
// Created by mark on 2021/3/1.
//


#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int main() {
    hipDeviceProp_t deviceProp{};
    int count = 0;
    hipError_t error;
    if(hipGetDeviceCount(&count)!=hipSuccess || count <= 0) {
        if(count >0)
            cout << hipGetErrorString(error) << endl;
        else
            cout << "no error but count = 0" << endl;
        exit(EXIT_FAILURE);
    }
    hipGetDeviceProperties(&deviceProp, 0);
    cout << deviceProp.computePreemptionSupported << endl;
    return 0;
}