//
// Created by root on 2021/3/3.
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#define DATA_SIZE 10000000

__global__ void vecAdd(int *a, int *b, int *c, int num) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int skip = gridDim.x * blockDim.x;
    for(int i = id; i < num; i+= skip) {
        c[i] = a[i] + b[i];
    }
}

__global__ void vecAbstract(int *a, int *b, int *c, int num) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int skip = gridDim.x * blockDim.x;
    for(int i = id; i < num; i+= skip) {
        c[i] = a[i] - b[i];
    }
}

void correctCheck(int *h, int type) {
    switch (type) {
        case 0:{
            for(int i = 0; i < DATA_SIZE; i ++)
            {
                if(h[i] != 0x1010101)
                {
                    printf("abstract fail at %d value: %d", i, h[i]);
                    return;
                }
            }
        }
        break;
        case 1: {
            for(int i = 0; i < DATA_SIZE; i ++)
            {
                if(h[i] != 0x3030303)
                {
                    printf("add fail at %d value: %d", i, h[i]);
                    return;
                }
            }
        }
    }
}

int main() {
    int nstreams = 2;
    hipSetDevice(0);
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync | hipDeviceMapHost);
    auto streams = (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++ ){
        hipStreamCreate(&streams[i]);
    }

    hipEvent_t start_event, stop_event;
    hipEventCreateWithFlags(&start_event, hipEventBlockingSync);
    hipEventCreateWithFlags(&stop_event, hipEventBlockingSync);

    float time;
    int * d_a, *d_b, *d_c_add, *d_c_abstract;
    int * h_a;
    // allocate memory
    hipHostMalloc(&h_a, DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipMalloc(&d_a, DATA_SIZE * sizeof(int));
    hipMalloc(&d_b, DATA_SIZE * sizeof(int));
    hipMalloc(&d_c_add, DATA_SIZE * sizeof(int));
    hipMalloc(&d_c_abstract, DATA_SIZE * sizeof(int));
    hipMemset(d_a, 0x2, DATA_SIZE * sizeof(int));
    hipMemset(d_b, 0x1, DATA_SIZE * sizeof(int));

    dim3 blockDim(512);
    dim3 gridDim(1);
    hipEventRecord(start_event, 0);
    vecAbstract<<<gridDim, blockDim, 0, streams[0]>>>(d_a, d_b, d_c_abstract, DATA_SIZE);
    vecAdd<<<gridDim, blockDim, 0, streams[1]>>>(d_a, d_b, d_c_add, DATA_SIZE);

    hipMemcpyAsync(h_a, d_c_abstract, sizeof(int) * DATA_SIZE, hipMemcpyDeviceToHost, streams[0]);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&time, start_event, stop_event);
    printf("vecabstract cost: %.2f \n", time);
    hipStreamSynchronize(streams[0]);
    correctCheck(h_a, 0);
    hipMemcpyAsync(h_a, d_c_add, sizeof(int) * DATA_SIZE, hipMemcpyDeviceToHost, streams[1]);
    hipDeviceSynchronize();
    correctCheck(h_a, 1);
}